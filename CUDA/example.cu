
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
	printf("Hello cuda from thread %d\n", i);
    b[i] = 2*a[i];
}

int main() {
    int ha[N], hb[N];
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    add<<<N, 1>>>(da, db);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipFree(da);
    hipFree(db);

    return 0;
}
