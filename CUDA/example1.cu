#include "hip/hip_runtime.h"
#include <iostream> 
#define BLOCK_SIZE 16
#define GRID_SIZE 1


__global__ void test(int *A, int *B,int *C)
{

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    //if (i < BLOCK_SIZE && j < BLOCK_SIZE)
        C[i][j] = A[i][j] + B[i][j];

}

int main()
{
	
	
	
    int* d_A;
    int* d_B;
    int* d_C;
	int A[BLOCK_SIZE][BLOCK_SIZE];
	int B[BLOCK_SIZE][BLOCK_SIZE];
    int C[BLOCK_SIZE][BLOCK_SIZE];

    for(int i=0;i<BLOCK_SIZE;i++)
      for(int j=0;j<BLOCK_SIZE;j++)
      {
        A[i][j]=i+j;
        B[i][j]=i+j;
      }
	hipMalloc((void **)&d_A, sizeof A);
	hipMalloc((void **)&d_B, sizeof B);
	hipMalloc((void **)&d_C, sizeof C);
	hipMemcpy(d_A, A,sizeof A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B,sizeof B, hipMemcpyHostToDevice);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid(GRID_SIZE, GRID_SIZE); 

    test<<<dimGrid, dimBlock>>>(d_A,d_B,d_C); 

    hipMemcpy(C,d_C,sizeof C , hipMemcpyDeviceToHost);

    for(int i=0;i<BLOCK_SIZE;i++)
      for(int j=0;j<BLOCK_SIZE;j++)
      {
        printf("%d\n",C[i][j]);

      }
}